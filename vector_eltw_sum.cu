
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function to add two vectors
__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 512;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // Allocate vectors in device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel using 1 block of 512 threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < N; i++) {
        printf("h_C[%d] = %f\n", i, h_C[i]);
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            printf("Error: %f + %f != %f\n", h_A[i], h_B[i], h_C[i]);
            break;
        }
    }
    printf("Vector addition completed successfully!\n");

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
